#include "hip/hip_runtime.h"

#include <stdio.h>
#include <chrono>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>


#define USBT_ZONE_SIZE			1000.f
#define USBT_ZONE_DIMENSION		65

typedef unsigned char TDensityVal;
typedef unsigned short TMaterialId;

#include "CudaVdGenerator.h"

//====================================================================================
// DLL declarations
//====================================================================================

extern "C" __declspec(dllexport) int CudaGetInfo(void);
extern "C" __declspec(dllexport) int CudaGenerateVd(TVdGenBlock* vdGenBlock);


//====================================================================================
// cuda perlin noise
//====================================================================================

__device__ const int p[512] = { 151, 160, 137, 91, 90, 15, 131, 13, 201, 95, 96, 53, 194, 233, 7, 225, 140, 36, 103, 30, 69, 142, 8, 99, 37, 240, 21, 10, 23, 190, 6, 148, 247, 120, 234, 75, 0, 26, 197, 62, 94, 252, 219, 203, 117, 35, 11, 32, 57, 177, 33, 88, 237, 149, 56, 87, 174, 20, 125, 136, 171, 168, 68, 175, 74, 165, 71, 134, 139, 48, 27, 166, 77, 146, 158, 231, 83, 111, 229, 122, 60, 211, 133, 230, 220, 105, 92, 41, 55, 46, 245, 40, 244, 102, 143, 54, 65, 25, 63, 161, 1, 216, 80, 73, 209, 76, 132, 187, 208, 89, 18, 169, 200, 196, 135, 130, 116, 188, 159, 86, 164, 100, 109, 198, 173, 186, 3, 64, 52, 217, 226, 250, 124, 123, 5, 202, 38, 147, 118, 126, 255, 82, 85, 212, 207, 206, 59, 227, 47, 16, 58, 17, 182, 189, 28, 42, 223, 183, 170, 213, 119, 248, 152, 2, 44, 154, 163, 70, 221, 153, 101, 155, 167, 43, 172, 9, 129, 22, 39, 253, 19, 98, 108, 110, 79, 113, 224, 232, 178, 185, 112, 104, 218, 246, 97, 228, 251, 34, 242, 193, 238, 210, 144, 12, 191, 179, 162, 241, 81, 51, 145, 235, 249, 14, 239, 107, 49, 192, 214, 31, 181, 199, 106, 157, 184, 84, 204, 176, 115, 121, 50, 45, 127, 4, 150, 254, 138, 236, 205, 93, 222, 114, 67, 29, 24, 72, 243, 141, 128, 195, 78, 66, 215, 61, 156, 180, 151, 160, 137, 91, 90, 15, 131, 13, 201, 95, 96, 53, 194, 233, 7, 225, 140, 36, 103, 30, 69, 142, 8, 99, 37, 240, 21, 10, 23, 190, 6, 148, 247, 120, 234, 75, 0, 26, 197, 62, 94, 252, 219, 203, 117, 35, 11, 32, 57, 177, 33, 88, 237, 149, 56, 87, 174, 20, 125, 136, 171, 168, 68, 175, 74, 165, 71, 134, 139, 48, 27, 166, 77, 146, 158, 231, 83, 111, 229, 122, 60, 211, 133, 230, 220, 105, 92, 41, 55, 46, 245, 40, 244, 102, 143, 54, 65, 25, 63, 161, 1, 216, 80, 73, 209, 76, 132, 187, 208, 89, 18, 169, 200, 196, 135, 130, 116, 188, 159, 86, 164, 100, 109, 198, 173, 186, 3, 64, 52, 217, 226, 250, 124, 123, 5, 202, 38, 147, 118, 126, 255, 82, 85, 212, 207, 206, 59, 227, 47, 16, 58, 17, 182, 189, 28, 42, 223, 183, 170, 213, 119, 248, 152, 2, 44, 154, 163, 70, 221, 153, 101, 155, 167, 43, 172, 9, 129, 22, 39, 253, 19, 98, 108, 110, 79, 113, 224, 232, 178, 185, 112, 104, 218, 246, 97, 228, 251, 34, 242, 193, 238, 210, 144, 12, 191, 179, 162, 241, 81, 51, 145, 235, 249, 14, 239, 107, 49, 192, 214, 31, 181, 199, 106, 157, 184, 84, 204, 176, 115, 121, 50, 45, 127, 4, 150, 254, 138, 236, 205, 93, 222, 114, 67, 29, 24, 72, 243, 141, 128, 195, 78, 66, 215, 61, 156, 180 };

__device__ float fade(float t) { return t * t * t * (t * (t * 6 - 15) + 10); }

__device__ float lerp(float t, float a, float b) { return a + t * (b - a); }

__device__ float grad(int hash, float x, float y, float z) {
    int h = hash & 15;
    float u = h < 8 ? x : y,
        v = h < 4 ? y : h == 12 || h == 14 ? x : z;
    return ((h & 1) == 0 ? u : -u) + ((h & 2) == 0 ? v : -v);
}
__device__ float noise(float x, float y, float z) {
    int X = (int)floor(x) & 255;
    int Y = (int)floor(y) & 255;
    int Z = (int)floor(z) & 255;

    x -= floor(x);
    y -= floor(y);
    z -= floor(z);

    float u = fade(x);
    float v = fade(y);
    float w = fade(z);

    int A = p[X] + Y, AA = p[A] + Z, AB = p[A + 1] + Z;
    int B = p[X + 1] + Y, BA = p[B] + Z, BB = p[B + 1] + Z;

    return lerp(w, lerp(v, lerp(u, grad(p[AA], x, y, z),
        grad(p[BA], x - 1, y, z)),
        lerp(u, grad(p[AB], x, y - 1, z),
            grad(p[BB], x - 1, y - 1, z))),
        lerp(v, lerp(u, grad(p[AA + 1], x, y, z - 1),
            grad(p[BA + 1], x - 1, y, z - 1)),
            lerp(u, grad(p[AB + 1], x, y - 1, z - 1),
                grad(p[BB + 1], x - 1, y - 1, z - 1))));
}

//====================================================================================
// 
//====================================================================================

__device__ float3 operator+(const float3& a, const float3& b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 voxelIndexToRelPos(const int x, const int y, const int z, int vd_num, float vd_size) {
    const float step = vd_size / (vd_num - 1);
    const float s = -vd_size / 2;
    float3 v = make_float3(s, s, s);
    float3 a = make_float3(x * step, y * step, z * step);
    v = v + a;
    return v;
}

__device__ float clcGroundLevel(float3 v, int vd_num, float vd_size) {
    const float scale1 = 0.001f; // small
    const float scale2 = 0.0004f; // medium
    const float scale3 = 0.00009f; // big

    const float noise_small = noise(v.x * scale1, v.y * scale1, 0) * 0.5f;
    const float noise_medium = noise(v.x * scale2, v.y * scale2, 0) * 5;
    const float noise_big = noise(v.x * scale3, v.y * scale3, 0) * 10;
    const float gl = ((noise_small + noise_medium + noise_big) * 100) + 205.f;

    return gl;
}

__device__ float clcDensityByGroundLevel(float3 v, float gl) {
    const float Z = v.z;
    const float D = Z - gl;

    if (D > 500) {
        return 0.f;
    }

    if (D < -500) {
        return 1.f;
    }

    const float density = 1 - (1 / (1 + exp(-(Z-gl) / 20)));
    return density;
}

__device__ int clcLinearIndex(int x, int y, int z, int n) {
    return x * n * n + y * n + z;
};


__global__ void cudaGenerateZoneVd(TZoneData* devZoneData, TDensityVal* devVd, TMaterialId* devMd, int vd_num, float vd_size, int totalZoneCount) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= totalZoneCount) {
        return;
    }

    TZoneData zd = devZoneData[idx];
    auto t = vd_num * vd_num * vd_num * idx;

    TDensityVal* vd = devVd + t;
    TMaterialId* md = devMd + t;

    const float3 zoneOrigin = make_float3(zd.x, zd.y, zd.z);
    const float step = vd_size / (vd_num - 1);
    const float s = -vd_size / 2;

    int i = 0;
    for (int x = 0; x < vd_num; x++) {
        for (int y = 0; y < vd_num; y++) {
            float3 v0 = voxelIndexToRelPos(x, y, 0, vd_num, vd_size);
            v0 = v0 + zoneOrigin;
            const float gl = clcGroundLevel(v0, vd_num, vd_size);

            for (int z = 0; z < vd_num; z++) {              
                float3 v = voxelIndexToRelPos(x, y, z, vd_num, vd_size);
                v = v + zoneOrigin;

                float density = clcDensityByGroundLevel(v, gl);
                int d = 255 * density;

                vd[i] = d;
                md[i] = 1;

                i++;
            }

        }
    } 
}

#define LOD_ARRAY_SIZE 7

__global__ void cudaMakeCache(TZoneData* devZoneData, TDensityVal* devVd, int* devCd, int vd_num, float vd_size, int totalZoneCount) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= totalZoneCount) {
        return;
    }

    //TZoneData zd = devZoneData[idx];
    TZoneData* zd_p = &devZoneData[idx];
    auto t = vd_num * vd_num * vd_num * idx;

    TDensityVal* vd = devVd + t;
    int* cd = devCd + t;

    for (int lod = 0; lod < LOD_ARRAY_SIZE; lod++) {
        zd_p->c[lod] = 0;
    }

    for (int x = 0u; x < vd_num; x++) {
        for (int y = 0u; y < vd_num; y++) {
            for (int z = 0u; z < vd_num; z++) {

                int offset = 0;

                for (int lod = 0; lod < LOD_ARRAY_SIZE; lod++) {
                    int s = 1 << lod;

                    int n = (vd_num - 1) >> lod;

                    if (x >= s && y >= s && z >= s) {
                        if (x % s == 0 && y % s == 0 && z % s == 0) {
                            int li[8];
                            li[7] = clcLinearIndex(x, y - s, z, vd_num);
                            li[6] = clcLinearIndex(x, y, z, vd_num);
                            li[5] = clcLinearIndex(x - s, y - s, z, vd_num);
                            li[4] = clcLinearIndex(x - s, y, z, vd_num);
                            li[3] = clcLinearIndex(x, y - s, z - s, vd_num);
                            li[2] = clcLinearIndex(x, y, z - s, vd_num);
                            li[1] = clcLinearIndex(x - s, y - s, z - s, vd_num);
                            li[0] = clcLinearIndex(x - s, y, z - s, vd_num);


                            char corner[8];
                            for (auto i = 0; i < 8; i++) {
                                corner[i] = (vd[li[i]] <= 127) ? -127 : 0;
                            }

                            unsigned long caseCode = ((corner[0] >> 7) & 0x01)
                                | ((corner[1] >> 6) & 0x02)
                                | ((corner[2] >> 5) & 0x04)
                                | ((corner[3] >> 4) & 0x08)
                                | ((corner[4] >> 3) & 0x10)
                                | ((corner[5] >> 2) & 0x20)
                                | ((corner[6] >> 1) & 0x40)
                                | (corner[7] & 0x80);

                            if (caseCode != 0 && caseCode != 255) {
                                cd[offset + zd_p->c[lod]] = li[1];
                                zd_p->c[lod]++;
                            }
                        }
                    }

                    offset += n * n * n;
                }


            }
        }
    }
    

    
}

//====================================================================================
// 
//====================================================================================

typedef unsigned __int64 uint64;

uint64 time_ms() {
    return std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
}

int CudaGetInfo(void) {
    return 0;
}

int CudaGenerateVd(TVdGenBlock* vdGenBlock) {
    hipError_t err = hipSuccess;

    if (vdGenBlock == nullptr) {
        return -1;
    }

    uint64 start0 = time_ms();

    int vd_num = USBT_ZONE_DIMENSION;
    float vd_size = USBT_ZONE_SIZE;

    printf("total_zone_count -> %d \n", vdGenBlock->size);

    auto start1 = time_ms();

    TDensityVal* d_vd = NULL;
    err = hipMalloc((void**)&d_vd, vdGenBlock->vd_size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device TDensityVal (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    TMaterialId* d_md = NULL;
    err = hipMalloc((void**)&d_md, vdGenBlock->md_size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device TMaterialId (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    TZoneData* d_zd = NULL;
    err = hipMalloc((void**)&d_zd, vdGenBlock->zd_size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device TZoneData (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    int* d_cd = NULL;
    err = hipMalloc((void**)&d_cd, vdGenBlock->cd_size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device cach data (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_zd, vdGenBlock->zoneData, vdGenBlock->zd_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy zone data from host to device (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    auto end1 = time_ms();
    printf("%d ms\n", (int)(end1 - start1));

    int threadsPerBlock = 256;
    int blocksPerGrid = (vdGenBlock->size + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    auto t1 = time_ms();
    cudaGenerateZoneVd << <blocksPerGrid, threadsPerBlock >> > (d_zd, d_vd, d_md, vd_num, vd_size, vdGenBlock->size);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch cudaGenerateZoneVd kernel (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }


    cudaMakeCache << <blocksPerGrid, threadsPerBlock >> > (d_zd, d_vd, d_cd, vd_num, vd_size, vdGenBlock->size);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch cudaMakeCache kernel (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    err = hipDeviceSynchronize();
    auto t2 = time_ms();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to hipDeviceSynchronize: %s \n", hipGetErrorString(err));
        return -1;
    } else {
        printf("%d ms\n", (int)(t2 - t1));
    }

    printf("Copy output data from the CUDA device to the host memory\n");
    printf("%d\n", vdGenBlock->vd_size);

    auto start2 = time_ms();

    err = hipMemcpy(vdGenBlock->voxelData, d_vd, vdGenBlock->vd_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(vdGenBlock->materialData, d_md, vdGenBlock->md_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(vdGenBlock->zoneData, d_zd, vdGenBlock->zd_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy zone data from device to host (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(vdGenBlock->cacheData, d_cd, vdGenBlock->cd_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy cache data from device to host (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    auto end2 = time_ms();
    printf("%d ms\n", (int)(end2 - start2));

    // Free device global memory
    err = hipFree(d_vd);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    err = hipFree(d_md);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    err = hipFree(d_zd);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        return -1;
    }

    auto end0 = time_ms();
    printf("Finished: %d ms\n", (int)(end0 - start0));

    return 0;
}

